/* implemantation of the bubble-sort algorithm in CUDA */

#include <hip/hip_runtime.h>
#include <stdio.h>


#ifdef _WIN32
using uint = unsigned int;
#endif

#define BLOCK_SIZE 256

uint bubblesort(float*, unsigned long long);
hipError_t cuda_bubblesort(float*, unsigned long long);
__host__ __device__ void swap(float &, float &);  
__global__ void bubble_kernel1(float*, unsigned long long, unsigned short int*);
__global__ void bubble_kernel2(float*, unsigned long long, unsigned short int*);

////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char *argv[])
{
  unsigned long long size = 3000;
  
  float *h_array = (float*)malloc(size*sizeof(float));
  for(unsigned long long i = 0; i < size; ++i) h_array[i] = float(size-i);

  cuda_bubblesort(h_array, size);
  //bubblesort(h_array, size);

  //for(uint i = 0; i < size; ++i)
  //{
    //printf("element %d: %f\n", i, h_array[i]);
  //}

  free(h_array);
  
  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ __device__
void swap(float &x, float &y)
{
  x = x + y;
  y = x - y;
  x = x - y;
}

uint bubblesort(float *h_array, unsigned long long size)
{
  uint newn, counter = 0;

  do
  {
    newn = 1;
    for(uint i = 0; i < size-1; ++i)
    {
      if(h_array[i] > h_array[i+1])
      {
	      swap(h_array[i], h_array[i+1]);
	      newn = i+1;
      }
    }
    
    size = newn;
    
    ++counter;
    
  } while(size > 1);

  printf("sorted after %d steps.\n", counter);
  
  return 0;
}

hipError_t cuda_bubblesort(float *h_array, unsigned long long size)
{
  hipError_t error;
  hipStream_t stream;
  hipStreamCreate(&stream);
  
  float *d_array;
  error = hipMalloc(&d_array, size*sizeof(float));
	     
  error = hipMemcpyAsync(d_array, h_array, size*sizeof(float), hipMemcpyHostToDevice, stream);


  /////////////////////////////////////////

  unsigned long long number_of_blocks = ( ((size+1)/2) + BLOCK_SIZE - 1 ) / BLOCK_SIZE;
  
  uint *h_swapped = (uint*)malloc(sizeof(uint));
  unsigned short int *d_swapped;
  error = hipMalloc(&d_swapped, sizeof(unsigned short int));

  unsigned long long counter = 0;
  
  do
  {
    h_swapped[0] = 0;

    error = hipMemcpyAsync(d_swapped, h_swapped, sizeof(unsigned short int),
			    hipMemcpyHostToDevice, stream);

    bubble_kernel1<<<number_of_blocks, BLOCK_SIZE, 0, stream>>>(d_array, size, d_swapped);
    bubble_kernel2<<<number_of_blocks, BLOCK_SIZE, 0, stream>>>(d_array, size, d_swapped);
    
    error = hipMemcpyAsync(h_swapped, d_swapped, sizeof(unsigned short int),
			    hipMemcpyDeviceToHost, stream);

    ++counter;
  } while(h_swapped[0]);
  
  /////////////////////////////////////////

  hipDeviceSynchronize();
  error = hipMemcpy(h_array, d_array, size*sizeof(float), hipMemcpyDeviceToHost);

  error = hipFree(d_array);
  
  printf("sorted after %lld steps.\n", counter);
	     
  return error;
}

__global__
void bubble_kernel1(float *d_array, unsigned long long size, unsigned short int *d_swapped)
{
  unsigned long long i = blockIdx.x*blockDim.x + threadIdx.x;

  if(2*i < size)
  {
  if(d_array[2*i] > d_array[2*i+1])
  {
    swap(d_array[2*i], d_array[2*i+1]); 
    d_swapped[0] = 1;
  }
  }
}

__global__
void bubble_kernel2(float *d_array, unsigned long long size, unsigned short int *d_swapped)
{
  unsigned long long i = blockIdx.x*blockDim.x + threadIdx.x;

  if(2*i+1 < size-1)
  {
  if(d_array[2*i+1] > d_array[2*i+2])
  {
    swap(d_array[2*i+1], d_array[2*i+2]);
    d_swapped[0] = 1;
  }
  }
}
