#include "hip/hip_runtime.h"
/* a simple demonstration of multi-device management with CUDA and MPI */
//compile: nvcc -O3 -o cluster    -I /usr/lib/x86_64-linux-gnu/openmpi/include/ cluster.cu -L /usr/lib/x86_64-linux-gnu/openmpi/lib/ -lmpi
//execute: mpiexec -H <name1>,<name2> ./cluster

#include <mpi.h>
#include <stdio.h>

#define BLOCK_SIZE 192

__global__ void multiply(const double*, const double*, double*, uint);

int main(int argc, char** argv)
{
    MPI_Init(&argc, &argv);

    
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    int world_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

    char processor_name[MPI_MAX_PROCESSOR_NAME];
    int name_len;
    MPI_Get_processor_name(processor_name, &name_len);

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    hipDeviceProp_t deviceProp;
    for(uint device = 0; device < deviceCount; ++device)
    {
        hipGetDeviceProperties(&deviceProp, device);
        printf("Host %s with rank %d (of %d) has %d device(s). Its device #%d is %s and has compute capability %d.%d.\n", processor_name, world_rank, world_size, deviceCount, device+1, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
    
    uint size = 100*BLOCK_SIZE*world_size;

    double *v1  = (double*)malloc(size * sizeof(double));
    double *v2  = (double*)malloc(size * sizeof(double));
    double *res = (double*)malloc(size * sizeof(double));
    
    for(uint i = 0; i < size; ++i)
    {
      v1[i] = 0.0001*i;
      v2[i] = 0.0002*i;
    }
    
    double* dv1;
    double* dv2;
    double* dres;
    hipMalloc(&dv1,  size*sizeof(double));
    hipMalloc(&dv2,  size*sizeof(double));
    hipMalloc(&dres, size*sizeof(double));   

    hipMemcpy(dv1, v1, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dv2, v2, size*sizeof(double), hipMemcpyHostToDevice);
    
    multiply<<<(size + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE>>>(dv1,dv2,dres,size);
    
    hipMemcpy(res, dres, size*sizeof(double), hipMemcpyDeviceToHost);
   
    for(uint i = 0; i < size; ++i)
      printf("number %d: %f * %f = %f\n", world_rank*size+i, v1[i], v2[i], res[i]);    

    
    MPI_Finalize();

    return 0;
}

__global__ void multiply(const double *v1, const double *v2, double *res, uint size)
{
  uint index = blockIdx.x*blockDim.x + threadIdx.x;

  if(index < size)
      res[index] = v1[index] * v2[index];
}
