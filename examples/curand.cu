/* a demonstration of the usage of cuRAND (host and device API)*/

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>

#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>


#ifdef _WIN32
using uint = unsigned int;
#endif

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

__global__
void setup_kernel(hiprandState *state, uint n)
{
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(NULL, id, 0, &state[id]);
}

__global__
void generate_kernel(hiprandState *state, float *result, uint n)
{
  int id = threadIdx.x + blockIdx.x*blockDim.x;

  if(id < n)
  {
    hiprandState localState = state[id];

    float x = hiprand_uniform(&localState);
    /* use x in device */
    result[id] = x;

    state[id] = localState;
  }
}


int main(int argc, char **argv)
{
    size_t n = 10;
    uint block_size = 128;
    
    ////////////////////////////////////////////////////////////////////////////////////////
    
    float *hostResults = (float*)malloc(n*sizeof(float));
    float *devResults;
    hipMalloc(&devResults, n*sizeof(float));
    
    hiprandState *devStates;
    hipMalloc((void **)&devStates , n*sizeof(hiprandState));
    setup_kernel<<<(n+block_size-1)/block_size, block_size>>>(devStates, n);
    generate_kernel<<<(n+block_size-1)/block_size, block_size>>>(devStates, devResults, n);
    hipFree(devStates);

    hipMemcpy(hostResults, devResults, n*sizeof(float), hipMemcpyDeviceToHost);

    for(uint i = 0; i < n; i++) printf("%f\n", hostResults[i]);
    
    ////////////////////////////////////////////////////////////////////////////////////////
    printf("\n");
    
    hiprandGenerator_t gen;
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    long long seed = time(NULL);
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
    CURAND_CALL(hiprandGenerateUniform(gen, devResults, n));
    CURAND_CALL(hiprandDestroyGenerator(gen));

    hipMemcpy(hostResults, devResults, n*sizeof(float), hipMemcpyDeviceToHost);

    for(uint i = 0; i < n; i++) printf("%f\n", hostResults[i]);
    
    ////////////////////////////////////////////////////////////////////////////////////////
    
    hipFree(devResults);
    free(hostResults);
  
    return 0;
}
