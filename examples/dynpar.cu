/* Demonstration of dynamic parllelism */

#include <hip/hip_runtime.h>
#include <stdio.h>


#ifdef _WIN32
using uint = unsigned int;
#endif

__global__ void kernel_parent(float *data, uint N);
__global__ void kernel_child(float *data, uint N);

int main(int argc, char *argv[])
{
  uint N = 19200;
  float *host = (float *)malloc(N * sizeof(float));

  float *device;
  hipMalloc(&device, N * sizeof(float));
  hipMemcpy(device, host, N * sizeof(float), hipMemcpyHostToDevice);

  // spawns a kernel with one thread
  kernel_parent<<<1, 1>>>(device, N);

  hipError_t error = hipDeviceSynchronize();
  if (error == hipSuccess)
    return 0;
  else
  {
    printf(hipGetErrorString(error));
    return -1;
  }
}

/////////////////////////////////////////////////////////

__global__ void kernel_parent(float *data, uint N)
{
  uint newN = N / 2;
  if (newN == 0)
    return; //stops when each thread has exactly one point
  //spawn two new kernels with one thread each
  //copy half the data

  hipStream_t stream1;
  hipStreamCreateWithFlags(&stream1, hipStreamDefault);
  float *device1;
  hipMalloc(&device1, newN * sizeof(float));
  hipMemcpyAsync(device1, data, newN * sizeof(float), hipMemcpyDeviceToDevice, stream1);

  kernel_child<<<1, 1, 0, stream1>>>(device1, newN);

  hipStream_t stream2;
  hipStreamCreateWithFlags(&stream2, hipStreamDefault);
  float *device2;
  hipMalloc(&device2, newN * sizeof(float));
  hipMemcpyAsync(device2, data + newN, newN * sizeof(float), hipMemcpyDeviceToDevice, stream2);

  kernel_child<<<1, 1, 0, stream2>>>(device2, newN);
}

__global__ void kernel_child(float *data, uint N)
{
  uint newN = N / 2;
  if (newN == 0)
    return; //stops when each thread has exactly one point
  //spawn two new kernels with one thread each
  //copy half the data

  hipStream_t stream1;
  hipStreamCreateWithFlags(&stream1, hipStreamDefault);
  float *device1;
  hipMalloc(&device1, newN * sizeof(float));
  hipMemcpyAsync(device1, data, newN * sizeof(float), hipMemcpyDeviceToDevice, stream1);

  kernel_child<<<1, 1, 0, stream1>>>(device1, newN);

  hipStream_t stream2;
  hipStreamCreateWithFlags(&stream2, hipStreamDefault);
  float *device2;
  hipMalloc(&device2, newN * sizeof(float));
  hipMemcpyAsync(device2, data + newN, newN * sizeof(float), hipMemcpyDeviceToDevice, stream2);

  kernel_child<<<1, 1, 0, stream2>>>(device2, newN);
}
