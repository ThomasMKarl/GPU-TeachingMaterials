/* solving a dense system of linear equations with LU-decompositon in cuSOLVER */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <hipsolver.h>


void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++)
    {
        for(int col = 0 ; col < n ; col++)
	{
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}

int main(int argc, char*argv[])
{
/*       | 1 2 3  |
 *   A = | 4 5 6  |
 *       | 7 8 10 |
 *
 * without pivoting: A = L*U
 *       | 1 0 0 |      | 1  2  3 |
 *   L = | 4 1 0 |, U = | 0 -3 -6 |
 *       | 7 2 1 |      | 0  0  1 |
 *  
 * with pivoting: P*A = L*U
 *       | 0 0 1 |
 *   P = | 1 0 0 |
 *       | 0 1 0 |
 *
 *       | 1       0     0 |      | 7  8       10     |
 *   L = | 0.1429  1     0 |, U = | 0  0.8571  1.5714 |
 *       | 0.5714  0.5   1 |      | 0  0       -0.5   |
 */
    const int m = 3;
    const int lda = m;
    const int ldb = m;
    double A[lda*m] = {1.0, 4.0, 7.0, 2.0, 5.0, 8.0, 3.0, 6.0, 10.0};
    double B[m] = {1.0, 2.0, 3.0};
    double X[m]; /* X = A\B */
    double LU[lda*m]; /* L and U */
    int Ipiv[m];      /* host copy of pivoting sequence */
    int info;     /* host copy of error info */

//////////////////////////////////////////////////////////////////

/* step 1: create cusolver handle, bind a stream */
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverSetStream(cusolverH, stream);

/* step 2: copy A to device */
    double *d_A;
    hipMalloc(&d_A,    sizeof(double)*lda*m);
    double *d_B;
    hipMalloc(&d_B,    sizeof(double)*m);
    int *d_Ipiv;
    hipMalloc(&d_Ipiv, sizeof(int)*m);
    int *d_info;
    hipMalloc(&d_info, sizeof(int));

    hipMemcpy(d_A, A, sizeof(double)*lda*m, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(double)*m, hipMemcpyHostToDevice);
    
/* step 3: query working space of getrf */
    int lwork; /* size of workspace */
    double *d_work; /* device workspace for getrf */
    hipsolverDnDgetrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork);
    hipMalloc(&d_work, sizeof(double)*lwork);

/* step 4: LU factorization */
    const int pivot_on = 1;
    if (pivot_on) hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, d_Ipiv, d_info);
    else          hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, NULL,   d_info);
    hipDeviceSynchronize();

    if (pivot_on) hipMemcpy(Ipiv, d_Ipiv, sizeof(int)*m, hipMemcpyDeviceToHost);
    hipMemcpy(LU   , d_A   , sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    if(0 > info)
    {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    /*
 * step 5: solve A*X = B 
 *       | 1 |       | -0.3333 |
 *   B = | 2 |,  X = |  0.6667 |
 *       | 3 |       |  0      |
 *
 */
    if (pivot_on) hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, d_A, lda, d_Ipiv, d_B, ldb, d_info);
    else          hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, d_A, lda, NULL,   d_B, ldb, d_info);
    hipDeviceSynchronize();

    hipMemcpy(X , d_B, sizeof(double)*m, hipMemcpyDeviceToHost);

    printf("(L-1)+U =\n");
    printMatrix(m, m, LU, lda, "LU");
    printf("\nX =\n");
    printMatrix(m, 1, X, ldb, "X");
    printf("\n");
    

/* free resources */
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_Ipiv);
    hipFree(d_info);
    hipFree(d_work);

    hipsolverDnDestroy(cusolverH);
    hipStreamDestroy(stream);

    hipDeviceReset();

    return 0;
}
