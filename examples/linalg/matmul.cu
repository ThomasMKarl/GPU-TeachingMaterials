/* Matrix multiplication */
/* demonstrates shared memory */
/* demonstrates multi-dimensional blocks */

#include <hip/hip_runtime.h>
#include<stdio.h>


// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct
{
    uint width;
    uint height;
    float* elements;
    uint stride;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);
__global__ void MatMulKernelShared(const Matrix, const Matrix, Matrix);

// Get a matrix element
__device__
float GetElement(const Matrix A, uint row, uint col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__
void SetElement(Matrix A, uint row, uint col, float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__
Matrix GetSubMatrix(Matrix A, uint row, uint col) 
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C, uint shared = 0)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height; d_A.stride = A.width;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height; d_B.stride = B.width;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height; d_C.stride = C.width;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    
    if(shared) MatMulKernelShared<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    else MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__
void MatMulKernel(const Matrix A, const Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    uint row = blockIdx.y * blockDim.y + threadIdx.y;
    uint col = blockIdx.x * blockDim.x + threadIdx.x;
    for (uint e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e]
                * B.elements[e * B.width + col];
    
    C.elements[row * C.width + col] = Cvalue;
}

// Matrix multiplication kernel called by MatMul()
__global__
void MatMulKernelShared(const Matrix A, const Matrix B, Matrix C)
{    
    // Block row and column
    uint blockRow = blockIdx.y;
    uint blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    uint row = threadIdx.y;
    uint col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (uint m = 0; m < (A.width / BLOCK_SIZE); ++m)
    {
        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply Asub and Bsub together
        for (uint e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];
 
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}

int main(int argc, char *argv[])
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("#size device shared\n");
 
  
    Matrix A, B, C;

    for(uint n = 16; n <= 3200; n += 16)
    {

    A.width  = n; A.height = n;
    B.width  = n; B.height = n;
    C.width  = n; C.height = n;
  
    A.elements = (float*)malloc(n*n*sizeof(float));
    B.elements = (float*)malloc(n*n*sizeof(float));
    C.elements = (float*)malloc(n*n*sizeof(float));

    for(uint i = 0; i < n*n; ++i)
    {
        A.elements[i] = 1.0f; B.elements[i] = 2.0f;
    }
    
    hipEventRecord(start);
    MatMul(A, B, C, 0);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%d %f ", n, milliseconds);

    hipEventRecord(start);
    MatMul(A, B, C, 1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f\n", milliseconds);

    }

    /////////////////////////////////////////////////////

    
    /*for(uint i = 0; i < n*n; ++i)
    {
      printf("%f ", C.elements[i]);
    }
    printf("\n");*/
 
    return 0;  
}
