/* utilizing full duplex of PCIe */
/* demonstrates usage of streams */

#include <hip/hip_runtime.h>
#include<stdio.h>

int main()
{
  int *h_A, *h_B, *d_A, *d_B;
  
  printf("# MB msec.\n");
  for(unsigned long long N = 1000000; N < 1000000000; N += 50000000)
  {
    hipHostMalloc(&h_A, N*sizeof(int), hipHostMallocDefault);
    hipMalloc(&d_A, N*sizeof(int));
    hipHostMalloc(&h_B, N*sizeof(int), hipHostMallocDefault);
    hipMalloc(&d_B, N*sizeof(int));
    
    hipMemcpy(d_A, h_A, N*sizeof(int), hipMemcpyHostToDevice);
    
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  
  hipEventRecord(start);
  
    hipMemcpyAsync(d_B, h_B, N*sizeof(int),
		    hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(h_A, d_A, N*sizeof(int),
		    hipMemcpyDeviceToHost, stream2);

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0.0;
  hipEventElapsedTime(&milliseconds, start, stop);
  
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  
    printf("%f %f\n", 2*N*sizeof(int)/(1024.0f*1024.0f), milliseconds);

    hipFree(d_A);
    hipFree(d_B);
    hipHostFree(h_A);
    hipHostFree(h_B);
  }

  return 0;
}
