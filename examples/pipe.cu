#include "hip/hip_runtime.h"
/* Comparison of a normal vector addition with an asynchronous one, the latter is divided in smaller memory batches */
#include<stdio.h>
#include<gpuerror.h>

////////////////////////////////////////////////////////////////

int VecAdd      (float*, float*, float*, uint);
int VecAddAsync (float*, float*, float*, uint);
__global__ void cuVecAdd(float*, float*, float*, uint);
  
int main()
{
  cudaDeviceTest();
  
  uint N = 1280000;
  uint size = N*sizeof(float);
  
  float *h_A, *h_B, *h_C;
  CUDA_CALL(hipHostMalloc(&h_A, size));
  CUDA_CALL(hipHostMalloc(&h_B, size));
  CUDA_CALL(hipHostMalloc(&h_C, size));
  for(uint i = 0; i < N; ++i)
  {
    h_A[i] = 1.0f;
    h_B[i] = 2.0f;
  }

  CUDA_CALL(VecAdd(h_A, h_B, h_C, N));
  CUDA_CALL(VecAddAsync(h_A, h_B, h_C, N));
  for(uint i = 0; i < 1; ++i) printf("%f\n", h_C[12345]);

  CUDA_CALL(hipHostFree(h_A));
  CUDA_CALL(hipHostFree(h_B));
  CUDA_CALL(hipHostFree(h_C));

  return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////

int VecAdd(float *A, float *B, float *C, uint N)
{
  unsigned long long size = N*sizeof(float);
  
  float *d_A; CUDA_CALL(hipMalloc(&d_A, size));
  float *d_B; CUDA_CALL(hipMalloc(&d_B, size));
  float *d_C; CUDA_CALL(hipMalloc(&d_C, size));
  CUDA_CALL(hipMemcpyAsync(d_A, A, size, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_B, B, size, hipMemcpyHostToDevice));
  
  uint threadsPerBlock = 128;
  uint blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
  cuVecAdd<<<blocks,threadsPerBlock>>>(d_A, d_B, d_C, N);

  CUDA_CALL(hipMemcpyAsync(C, d_C, size, hipMemcpyDeviceToHost));
  CUDA_CALL(hipPeekAtLastError());
  #ifdef DEBUG_CUDA
    CUDA_CALL(hipDeviceSynchronize());
  #endif

  CUDA_CALL(hipFree(d_A));
  CUDA_CALL(hipFree(d_B));
  CUDA_CALL(hipFree(d_C));

  return EXIT_SUCCESS;
}

int VecAddAsync(float *A, float *B, float *C, uint N)
{
  uint threadsPerBlock = 128;
  uint stride = 10*threadsPerBlock;
  uint blocks = (stride + threadsPerBlock - 1) / threadsPerBlock;
  
  unsigned long long size = stride*sizeof(float);
  
  float *d_A; CUDA_CALL(hipMalloc(&d_A, size));
  float *d_B; CUDA_CALL(hipMalloc(&d_B, size));
  float *d_C; CUDA_CALL(hipMalloc(&d_C, size));
  
  //////////////////////////////////////////////////////////////
  
  uint num = (N - stride + 1) / stride;
  for(uint i = 0; i < num; ++i)
  {
    CUDA_CALL(hipMemcpyAsync(d_A, A+i*stride,
			      size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyAsync(d_B, B+i*stride,
			      size, hipMemcpyHostToDevice));
  
    cuVecAdd<<<blocks,threadsPerBlock>>>(d_A, d_B, d_C, stride);
    #ifdef DEBUG_CUDA
      CUDA_CALL(hipPeekAtLastError());
      CUDA_CALL(hipDeviceSynchronize());
    #endif

    CUDA_CALL(hipMemcpyAsync(C+i*stride, d_C,
			      size, hipMemcpyDeviceToHost));
  }

  CUDA_CALL(hipMemcpyAsync(d_A, A+num*stride,
			    size, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_B, B+num*stride,
			    size, hipMemcpyHostToDevice));
  
  cuVecAdd<<<blocks,threadsPerBlock>>>(d_A, d_B, d_C, N-num*stride);
  #ifdef DEBUG_CUDA
    CUDA_CALL(hipPeekAtLastError());
    CUDA_CALL(hipDeviceSynchronize());
  #endif

  CUDA_CALL(hipMemcpyAsync(C+num*stride, d_C,
			    size, hipMemcpyDeviceToHost));

  //////////////////////////////////////////////////////////////
  
  CUDA_CALL(hipFree(d_A));
  CUDA_CALL(hipFree(d_B));
  CUDA_CALL(hipFree(d_C));

  return EXIT_SUCCESS;
}

__global__
void cuVecAdd(float *A, float *B, float *C, uint N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < N) C[i] = A[i] + B[i];
}
