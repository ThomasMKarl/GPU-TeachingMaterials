/* Comparison of a normal vector addition with an asynchronous one, the latter is divided in smaller memory batches */

#include <hip/hip_runtime.h>
#include<stdio.h>


#ifdef _WIN32
using uint = unsigned int;
#endif

#define CUDA_CALL(x) do { hipError_t err = x; if(err!=hipSuccess) { \
    printf("'%s' at %s:%d\n",hipGetErrorString(x),__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

////////////////////////////////////////////////////////////////

int VecAdd      (float*, float*, float*, uint);
int VecAddAsync (float*, float*, float*, uint);
__global__ void cuVecAdd(float*, float*, float*, uint);
  
int main(int argc, char **argv)
{
  uint N = 1280000;
  uint size = N*sizeof(float);
  
  float *h_A, *h_B, *h_C;
  CUDA_CALL(hipHostMalloc(&h_A, size, hipHostMallocDefault, hipHostMallocDefault));
  CUDA_CALL(hipHostMalloc(&h_B, size, hipHostMallocDefault, hipHostMallocDefault));
  CUDA_CALL(hipHostMalloc(&h_C, size, hipHostMallocDefault, hipHostMallocDefault));
  for(uint i = 0; i < N; ++i)
  {
    h_A[i] = 1.0f;
    h_B[i] = 2.0f;
  }

  if(!VecAdd(h_A, h_B, h_C, N))
    return EXIT_FAILURE;
  if(!VecAddAsync(h_A, h_B, h_C, N))
    return EXIT_FAILURE;

  for(uint i = 0; i < 1; ++i) printf("%f\n", h_C[12345]);

  CUDA_CALL(hipHostFree(h_A));
  CUDA_CALL(hipHostFree(h_B));
  CUDA_CALL(hipHostFree(h_C));

  return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////

int VecAdd(float *A, float *B, float *C, uint N)
{
  unsigned long long size = N*sizeof(float);
  
  float *d_A; CUDA_CALL(hipMalloc(&d_A, size));
  float *d_B; CUDA_CALL(hipMalloc(&d_B, size));
  float *d_C; CUDA_CALL(hipMalloc(&d_C, size));
  CUDA_CALL(hipMemcpyAsync(d_A, A, size, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_B, B, size, hipMemcpyHostToDevice));
  
  uint threadsPerBlock = 128;
  uint blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
  cuVecAdd<<<blocks,threadsPerBlock>>>(d_A, d_B, d_C, N);

  CUDA_CALL(hipMemcpyAsync(C, d_C, size, hipMemcpyDeviceToHost));
  CUDA_CALL(hipPeekAtLastError());
  #ifdef DEBUG_CUDA
    CUDA_CALL(cudaDeviceSynchronize());
  #endif

  CUDA_CALL(hipFree(d_A));
  CUDA_CALL(hipFree(d_B));
  CUDA_CALL(hipFree(d_C));

  return EXIT_SUCCESS;
}

int VecAddAsync(float *A, float *B, float *C, uint N)
{
  uint threadsPerBlock = 128;
  uint stride = 10*threadsPerBlock;
  uint blocks = (stride + threadsPerBlock - 1) / threadsPerBlock;
  
  unsigned long long size = stride*sizeof(float);
  
  float *d_A; CUDA_CALL(hipMalloc(&d_A, size));
  float *d_B; CUDA_CALL(hipMalloc(&d_B, size));
  float *d_C; CUDA_CALL(hipMalloc(&d_C, size));
  
  //////////////////////////////////////////////////////////////
  
  uint num = (N - stride + 1) / stride;
  for(uint i = 0; i < num; ++i)
  {
    CUDA_CALL(hipMemcpyAsync(d_A, A+i*stride,
			      size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyAsync(d_B, B+i*stride,
			      size, hipMemcpyHostToDevice));
  
    cuVecAdd<<<blocks,threadsPerBlock>>>(d_A, d_B, d_C, stride);
    #ifdef DEBUG_CUDA
      CUDA_CALL(cudaPeekAtLastError());
      CUDA_CALL(cudaDeviceSynchronize());
    #endif

    CUDA_CALL(hipMemcpyAsync(C+i*stride, d_C,
			      size, hipMemcpyDeviceToHost));
  }

  CUDA_CALL(hipMemcpyAsync(d_A, A+num*stride,
			    size, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_B, B+num*stride,
			    size, hipMemcpyHostToDevice));
  
  cuVecAdd<<<blocks,threadsPerBlock>>>(d_A, d_B, d_C, N-num*stride);
  #ifdef DEBUG_CUDA
    CUDA_CALL(cudaPeekAtLastError());
    CUDA_CALL(cudaDeviceSynchronize());
  #endif

  CUDA_CALL(hipMemcpyAsync(C+num*stride, d_C,
			    size, hipMemcpyDeviceToHost));

  //////////////////////////////////////////////////////////////
  
  CUDA_CALL(hipFree(d_A));
  CUDA_CALL(hipFree(d_B));
  CUDA_CALL(hipFree(d_C));

  return EXIT_SUCCESS;
}

__global__
void cuVecAdd(float *A, float *B, float *C, uint N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < N) C[i] = A[i] + B[i];
}
